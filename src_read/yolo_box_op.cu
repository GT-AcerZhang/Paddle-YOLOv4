#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/detection/yolo_box_op.h"
#include "paddle/fluid/operators/math/math_function.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>


// input: 输入张量input的指针，形状是[bz, 255, 13, 13]
// imgsize: 输入张量img_size张量的指针，形状是[bz, 2]
// boxes:  输出张量boxes的指针
// scores: 输出张量scores的指针
// conf_thresh: 0.0
// anchors: 指针，指向[142, 110, 192, 243, 459, 401]
// n: 8  批大小
// h: 13 一列的格子数；格子行数
// w: 13 一行的格子数；格子列数
// an_num: 3 每个格子有几个预测框
// class_num: 80
// box_num: 3*13*13
// input_size: 32*13=416
// clip_bbox: True
// scale:
// bias: -0.5 * (scale - 1.)
__global__ void KeYoloBoxFw(const T* input, const int* imgsize, T* boxes,
                            T* scores, const float conf_thresh,
                            const int* anchors, const int n, const int h,
                            const int w, const int an_num, const int class_num,
                            const int box_num, int input_size, bool clip_bbox,
                            const float scale, const float bias) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  T box[4];

  // 遍历所有预测框，共bz*3*13*13个
  //           8 * 3*13*13
  for (; tid < n * box_num; tid += stride) {
    int grid_num = h * w;    // 这张特征图格子总数，比如13x13=169

    // 获得4个坐标[i, j, ?, k, l]。输入张量input的形状是[bz, 3, 85, 13, 13]，用来取预测框。
    int i = tid / box_num;   // batch_size维的下标，第几张图片。box_num=3*13*13
    int j = (tid % box_num) / grid_num;  // 预测框的下标
    int k = (tid % grid_num) / w;        // h的坐标，网格内的y坐标
    int l = tid % w;                     // w的坐标，网格内的x坐标

    // 输入张量input的形状是[bz, 3, 85, 13, 13]，那么同一个格子相邻两个预测框的步长 an_stride 就是 85*13*13
    int an_stride = (5 + class_num) * grid_num;
    int img_height = imgsize[2 * i];     // 原图的高
    int img_width = imgsize[2 * i + 1];  // 原图的宽



    // ========================== 1.获取置信位，过滤置信位 ==========================
    // 输入张量input的形状是[bz, 3, 85, 13, 13]
    // an_num 就是 3
    // an_stride 就是 85*13*13
    // grid_num 就是 13*13
    // 实际上输入张量input是一个一维数组（指针），所以将坐标[i, j, 4, k, l]转换成真实位置obj_idx
    int obj_idx = GetEntryIndex(i, j, k * w + l, an_num, an_stride, grid_num, 4);
    T conf = sigmoid<T>(input[obj_idx]);   // 置信位经过sigmoid()激活
    if (conf < conf_thresh) {    // 置信位数值低于conf_thresh，就过滤这个预测框
      // 刚才说到，boxes初始化为0，scores初始化为0，所以不会填写这个位置的分数，
      // 即这个位置的分数是0。但返回的boxes形状固定是[bz, 3, 13, 13, 4]，返回的scores形状固定是[bz, 3, 13, 13, 80]，
      // 并不会因为分数被过滤而受影响。这也很好理解，每张图片过滤后剩下的boxes数量是不同的（很大概率），数量不同的话，
      // 结果就很难用一个张量(bz, M, 4)表示，因为每张图片的M不同，不能对齐。不如索性让返回的boxes形状固定是[bz, 3, 13, 13, 4]。
      continue;
    }


    // ========================== 2.填写boxes ==========================
    // 实际上输入张量input是一个一维数组（指针），所以将坐标[i, j, 0, k, l]转换成真实位置box_idx
    int box_idx = GetEntryIndex(i, j, k * w + l, an_num, an_stride, grid_num, 0);

    // 将坐标为box_idx处的box解码，用那个公式。box作为返回值。
    GetYoloBox<T>(box, input, anchors, l, k, j, h, input_size, box_idx,
                  grid_num, img_height, img_width, scale, bias);

    // 输出boxes张量的形状是[bz, 3, 13, 13, 4]，实际上输出张量boxes是一个一维数组（指针），所以这里转换一下坐标。
    box_idx = (i * box_num + j * grid_num + k * w + l) * 4;

    // boxes，输出张量，形状是[bz, 3, 13, 13, 4]，实际上输出张量boxes是一个一维数组（指针）
    // box，T box[4]，一个长度为4的一维数组
    // box_idx，将box写入到boxes的box_idx处。写入的坐标格式是x1y1x2y2
    // clip_bbox，是否把x1y1x2y2限制在图片内。
    CalcDetectionBox<T>(boxes, box, box_idx, img_height, img_width, clip_bbox);



    // ========================== 3.填写scores ==========================
    // 实际上输入张量input是一个一维数组（指针），所以将坐标[i, j, 5, k, l]转换成真实位置label_idx
    int label_idx = GetEntryIndex(i, j, k * w + l, an_num, an_stride, grid_num, 5);

    // 输出scores张量的形状是[bz, 3, 13, 13, 80]，实际上输出张量scores是一个一维数组（指针），所以这里转换一下坐标。
    int score_idx = (i * box_num + j * grid_num + k * w + l) * class_num;


    // scores，输出张量，形状是[bz, 3, 13, 13, 80]，实际上输出张量scores是一个一维数组（指针）
    // input，输入张量，形状是[bz, 3, 85, 13, 13]，实际上输入张量input是一个一维数组（指针）
    // label_idx，把80位条件概率从input的label_idx处取出
    // score_idx，写进scores的score_idx处
    // class_num，80
    // conf，置信位，已经经过sigmoid()激活
    // grid_num，13*13，网格数
    CalcLabelScore<T>(scores, input, label_idx, score_idx, class_num, conf, grid_num);
    // boxes和scores，根据函数副作用返回
  }
}

// x = fluid.layers.data(name='x', shape=[255, 13, 13], dtype='float32')
// img_size = fluid.layers.data(name='img_size',shape=[2],dtype='int64')
// anchors = [142, 110, 192, 243, 459, 401]
// boxes, scores = fluid.layers.yolo_box(x=x, img_size=img_size, class_num=80, anchors=anchors,
//                                 conf_thresh=0.0, downsample_ratio=32)

template <typename T>
class YoloBoxOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input =    ctx.Input<Tensor>("X");     // [bz, 255, 13, 13]
    auto* img_size = ctx.Input<Tensor>("ImgSize");   // [bz, 2]
    auto* boxes =    ctx.Output<Tensor>("Boxes");    // [bz, -1, 4]
    auto* scores =   ctx.Output<Tensor>("Scores");   // [bz, -1, 80]

    auto anchors = ctx.Attr<std::vector<int>>("anchors");   //  [142, 110, 192, 243, 459, 401]
    int class_num = ctx.Attr<int>("class_num");   // 80
    float conf_thresh = ctx.Attr<float>("conf_thresh");   // 0.0
    int downsample_ratio = ctx.Attr<int>("downsample_ratio");  // 32
    bool clip_bbox = ctx.Attr<bool>("clip_bbox");   // True
    float scale = ctx.Attr<float>("scale_x_y");
    float bias = -0.5 * (scale - 1.);

    const int n = input->dims()[0];   // bz，批大小
    const int h = input->dims()[2];   // 13，格子行数
    const int w = input->dims()[3];   // 13，格子列数
    const int box_num = boxes->dims()[1];   // 3*13*13
    const int an_num = anchors.size() / 2;  // 3
    int input_size = downsample_ratio * h;  // 32*13=416

    auto& dev_ctx = ctx.cuda_device_context();
    int bytes = sizeof(int) * anchors.size();   // ?*6
    auto anchors_ptr = memory::Alloc(dev_ctx, sizeof(int) * anchors.size());
    int* anchors_data = reinterpret_cast<int*>(anchors_ptr->ptr());
    const auto gplace = BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace());
    const auto cplace = platform::CPUPlace();
    memory::Copy(gplace, anchors_data, cplace, anchors.data(), bytes,
                 dev_ctx.stream());

    const T* input_data = input->data<T>();   // 获得输入张量input的指针，形状是[bz, 255, 13, 13]
    const int* imgsize_data = img_size->data<int>();   // 获得输入张量img_size张量的指针，形状是[bz, 2]
    T* boxes_data = boxes->mutable_data<T>({n, box_num, 4}, ctx.GetPlace());   // 获得输出张量boxes的指针
    T* scores_data = scores->mutable_data<T>({n, box_num, class_num}, ctx.GetPlace());   // 获得输出张量scores的指针
    math::SetConstant<platform::CUDADeviceContext, T> set_zero;
    set_zero(dev_ctx, boxes, static_cast<T>(0));   // 将boxes初始化为0
    set_zero(dev_ctx, scores, static_cast<T>(0));  // 将scores初始化为0

    int grid_dim = (n * box_num + 512 - 1) / 512;   // 暂时不明白什么意思
    grid_dim = grid_dim > 8 ? 8 : grid_dim;   // 三个输出层，都是得8

    // boxes_data和scores_data，根据函数副作用返回
    KeYoloBoxFw<T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_data, imgsize_data, boxes_data, scores_data, conf_thresh,
        anchors_data, n, h, w, an_num, class_num, box_num, input_size,
        clip_bbox, scale, bias);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(yolo_box, ops::YoloBoxOpCUDAKernel<float>,
                        ops::YoloBoxOpCUDAKernel<double>);



